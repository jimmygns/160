#include "hip/hip_runtime.h"
#include <gputk.h>

#define BLOCK_SIZE 16

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      gpuTKLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this lab
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

  float Cvalue = 0;

//     // i is the index of the block
//   if (row >= numCRows || col >= numCColumns) {
//     return;
//   }
//   for (int i = 0; i < ceil((1.0*numAColumns)/BLOCK_SIZE); i++) {
//     if (row < numARows && i*BLOCK_SIZE + threadIdx.x < numAColumns) {
//       As[threadIdx.y][threadIdx.x] = A[row*numAColumns + i*BLOCK_SIZE + threadIdx.x];
//     } else {
//       As[threadIdx.y][threadIdx.x] = 0;
//     }
//     if (col < numBColumns && i*BLOCK_SIZE + threadIdx.y < numBRows) {
//       Bs[threadIdx.y][threadIdx.x] = B[(i*BLOCK_SIZE+threadIdx.y)*numBColumns + col];
//     } else {
//       Bs[threadIdx.y][threadIdx.x] = 0;
//     }
//     __syncthreads();
//     for (int j = 0; j < BLOCK_SIZE; j++) {
//       Cvalue += As[threadIdx.y][j] * Bs[j][threadIdx.x];
//     }
//     __syncthreads();
//   }
//   C[row*numCColumns + col] = Cvalue;
// }


  for (int i = 0; i < ceil((1.0*numAColumns)/BLOCK_SIZE); i++) {
    if (row < numARows && i * BLOCK_SIZE + threadIdx.x < numAColumns) {
      As[threadIdx.y][threadIdx.x] = A[row * numAColumns + i * BLOCK_SIZE + threadIdx.x];
    } else {
      As[threadIdx.y][threadIdx.x] = 0;
    }
    if (i * BLOCK_SIZE + threadIdx.y < numBRows && col < numBColumns) {
      Bs[threadIdx.y][threadIdx.x] = B[(i * BLOCK_SIZE + threadIdx.y) * numBColumns + col];
    } else {
      Bs[threadIdx.y][threadIdx.x] = 0;
    }
    __syncthreads();
    for (int j = 0; j < BLOCK_SIZE; j++) {
      Cvalue += As[threadIdx.y][j] * Bs[j][threadIdx.x];
    }
    __syncthreads();
  }
  if (row >= numCRows || col >= numCColumns) {
    return;
  }

  C[row * numCColumns + col] = Cvalue;
}


int main(int argc, char **argv) {
  gpuTKArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  gpuTKLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float),
             hipMemcpyHostToDevice);

  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid(ceil((1.0*numCColumns)/BLOCK_SIZE), ceil((1.0*numCRows)/BLOCK_SIZE), 1);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float),
             hipMemcpyDeviceToHost);

  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
